#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>

#define SIZE 1024

__global__ void VectorAdd(int *a, int *b, int *c, int n) {
  int i = threadIdx.x;
  if (i < n) {
    c[i] = a[i] + b[i];
  }
}

/*
".cu": "cd $dir && nvcc $fileName -o $fileNameWithoutExt -arch=sm_86 &&
$dir$fileNameWithoutExt",
*/
int main(void) {
  int *a, *b, *c;

  hipMallocManaged(&a, SIZE * sizeof(int)),
      hipMallocManaged(&b, SIZE * sizeof(int)),
      hipMallocManaged(&c, SIZE * sizeof(int));

  for (int i = 0; i < SIZE; i++) {
    a[i] = b[i] = i;
    c[i] = 0;
  }

  int threadsPerBlock = 256;
  int blocksPerGrid = (SIZE + threadsPerBlock - 1) / threadsPerBlock;

  // VectorAdd<<<1, SIZE>>>(a, b, c, SIZE); // <<<1, SIZE>>> WAS AN ERROR
  VectorAdd<<<blocksPerGrid, threadsPerBlock>>>(a, b, c, SIZE);

  // Kernel execution happens here

  hipDeviceSynchronize(); // Wait for kernel to finish

  for (int i = 0; i < 10; i++) {
    printf("c[%d] = %d\n", i, c[i]);
  }

  hipFree(a), hipFree(b), hipFree(c);
} // WORKING
