
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
// #include <cuda_runtime.h>

#define SIZE 1024

__global__ void VectorAdd(int *a, int *b, int *c, int n) {
  int i = blockIdx.x;
  if (i < n) {
    c[i] = a[i] + b[i];
  }
}

int main(void) {
  int *a, *b, *c;
  hipError_t err;

  // Allocate Unified Memory – accessible from CPU or GPU
  err = hipMallocManaged(&a, SIZE * sizeof(int));
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate memory for a (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  err = hipMallocManaged(&b, SIZE * sizeof(int));
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate memory for b (error code %s)!\n",
            hipGetErrorString(err));
    hipFree(a);
    exit(EXIT_FAILURE);
  }

  err = hipMallocManaged(&c, SIZE * sizeof(int));
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate memory for c (error code %s)!\n",
            hipGetErrorString(err));
    hipFree(a);
    hipFree(b);
    exit(EXIT_FAILURE);
  }

  // Initialize vectors
  for (int i = 0; i < SIZE; i++) {
    a[i] = i;
    b[i] = i;
    c[i] = 0;
  }

  // Set the kernel launch parameters
  //   int threadsPerBlock = 256;
  //   int blocksPerGrid = (SIZE + threadsPerBlock - 1) / threadsPerBlock;
  int threadsPerBlock = 1;
  int blocksPerGrid = SIZE;

  // Launch the kernel
  VectorAdd<<<blocksPerGrid, threadsPerBlock>>>(a, b, c, SIZE);

  // Check for any errors launching the kernel
  err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to launch VectorAdd kernel (error code %s)!\n",
            hipGetErrorString(err));
    hipFree(a);
    hipFree(b);
    hipFree(c);
    exit(EXIT_FAILURE);
  }

  // Wait for the GPU to finish
  err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    fprintf(stderr,
            "hipDeviceSynchronize returned error code %s after launching "
            "VectorAdd!\n",
            hipGetErrorString(err));
    hipFree(a);
    hipFree(b);
    hipFree(c);
    exit(EXIT_FAILURE);
  }

  // Print the result
  for (int i = 0; i < 10; i++) {
    printf("c[%d] = %d\n", i, c[i]);
  }

  // Free memory
  hipFree(a);
  hipFree(b);
  hipFree(c);

  return 0;
}
// ".cu": "cd $dir && nvcc $fileName -o $fileNameWithoutExt -arch=sm_86 &&
// $dir$fileNameWithoutExt",