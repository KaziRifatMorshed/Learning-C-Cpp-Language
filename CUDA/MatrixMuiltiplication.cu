#include <cstddef>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <hip/hip_runtime.h>
#include <stdio.h>

// void MatrixMulFunction(int m, int n, int k, float *A, float *B, float *C) {
//   for (int row = 0; row < m; row++) {
//     for (int col = 0; col < k; col++) {
//       float sum = 0;
//       for (int i = 0; i < n; i++) {
//         float a = A[row * n + i];
//         float b = B[i * k + col];
//         sum += a * b;
//       }
//       C[row * k + col] = sum;
//     }
//   }
// }

__global__ void MatrixMulFunction(int m, int n, int k, float *A, float *B,
                                  float *C) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if ((row < m) && (col < k)) {
    float sum = 0.0f;
    for (int i = 0; i < n; i++) {
      sum += A[row * m + i] * B[col + k * i];
    }
    C[row * n + col] = sum;
  }
}

int main(void) {

  float *d_A, *d_B, *d_C;
  size_t mem_size_for_matrix = 3 * 3 * sizeof(float);
  hipError_t e1 = hipMallocManaged(&d_A, mem_size_for_matrix);
  hipError_t e2 = hipMallocManaged(&d_B, mem_size_for_matrix);
  hipError_t e3 = hipMallocManaged(&d_C, mem_size_for_matrix);

  if (e1 != hipSuccess || e2 != hipSuccess || e3 != hipSuccess) {
    printf("error in file %s (and %d is a line)", __FILE__, __LINE__);
    printf("Mem allocation error: \"%s : %s\"", hipGetErrorName(e1),
           hipGetErrorString(e1));
    // return 1;
    exit(EXIT_FAILURE);
  }

  float h_A[3][3] = {{2, 3, 4}, {3, 5, 6}, {4, 5, 3}};
  float h_B[3][3] = {{1, 2, 1}, {-1, 2, 1}, {3, 2, 1}};
  float h_C[3][3];

  // copy from Host RAM to GPU RAM
  hipMemcpy(d_A, h_A, mem_size_for_matrix, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, mem_size_for_matrix, hipMemcpyHostToDevice);

  // define block size and grid size
  // dim3 threadsPerBlock(16, 16, 0);
  dim3 threadsPerBlock(16, 16);
  dim3 numBlocks(((3 + threadsPerBlock.x - 1) / threadsPerBlock.x),
                 (3 + threadsPerBlock.y - 1) / threadsPerBlock.y);

  // launch cuda kernel
  MatrixMulFunction<<<numBlocks, threadsPerBlock>>>(3, 3, 3, (float *)d_A,
                                                    (float *)d_B, (float *)d_C);

  hipDeviceSynchronize();
  hipMemcpy(h_C, d_C, mem_size_for_matrix, hipMemcpyDeviceToHost);

  hipFree(d_A), hipFree(d_B), hipFree(d_C);

  printf("Matrix Multiplication in CUDA C\nPrinting Result:\n");
  for (int i = 0; i < 3; i++) {
    for (int j = 0; j < 3; j++) {
      printf("%2.2f\t", h_C[i][j]);
    }
    printf("\n");
  }
} // DONE
/*
Result
11 18 9
16 28 14
8 24 12
*/

/*
The provided code has a few issues that need to be corrected for it to work
correctly with CUDA. Here's the revised version:

1. The memory needs to be allocated on the GPU.
2. The memory needs to be copied to the GPU before the kernel launch.
3. The memory needs to be copied back from the GPU after the kernel execution.
4. Correcting the indexing in the kernel function.

Here's the corrected code:

```c
#include <cuda_runtime.h>
#include <stdio.h>

__global__ void MatrixMulFunction(int m, int n, int k, float *A, float *B, float
*C) { int row = blockIdx.y * blockDim.y + threadIdx.y; int col = blockIdx.x *
blockDim.x + threadIdx.x;

  if ((row < m) && (col < k)) {
    float sum = 0.0f;
    for (int i = 0; i < n; i++) {
      sum += A[row * n + i] * B[i * k + col];
    }
    C[row * k + col] = sum;
  }
}

int main(void) {
  int m = 3, n = 3, k = 3;
  float A[3][3] = {{2, 3, 4}, {3, 5, 6}, {4, 5, 3}};
  float B[3][3] = {{1, 2, 1}, {-1, 2, 1}, {3, 2, 1}};
  float C[3][3] = {0};

  float *d_A, *d_B, *d_C;
  size_t size = m * n * sizeof(float);

  // Allocate memory on the GPU
  cudaMalloc(&d_A, size);
  cudaMalloc(&d_B, size);
  cudaMalloc(&d_C, size);

  // Copy matrices A and B to the GPU
  cudaMemcpy(d_A, A, size, cudaMemcpyHostToDevice);
  cudaMemcpy(d_B, B, size, cudaMemcpyHostToDevice);

  // Define the block size and grid size
  dim3 threadsPerBlock(16, 16);
  dim3 numBlocks((k + threadsPerBlock.x - 1) / threadsPerBlock.x,
                 (m + threadsPerBlock.y - 1) / threadsPerBlock.y);

  // Launch the kernel
  MatrixMulFunction<<<numBlocks, threadsPerBlock>>>(m, n, k, d_A, d_B, d_C);

  // Copy the result matrix C back to the host
  cudaMemcpy(C, d_C, size, cudaMemcpyDeviceToHost);

  // Free GPU memory
  cudaFree(d_A);
  cudaFree(d_B);
  cudaFree(d_C);

  // Print the result
  printf("Matrix Multiplication in CUDA C\nPrinting Result:\n");
  for (int i = 0; i < m; i++) {
    for (int j = 0; j < k; j++) {
      printf("%2.2f\t", C[i][j]);
    }
    printf("\n");
  }

  return 0;
}
```

### Explanation:

1. **Memory Allocation**: The matrices `A`, `B`, and `C` are allocated on the
GPU using `cudaMalloc`.
2. **Memory Copy to GPU**: The matrices `A` and `B` are copied from the host to
the GPU using `cudaMemcpy`.
3. **Kernel Execution**: The kernel is launched with a grid and block
configuration.
4. **Memory Copy to Host**: The result matrix `C` is copied back from the GPU to
the host using `cudaMemcpy`.
5. **Free GPU Memory**: The allocated GPU memory is freed using `cudaFree`.

This code should now correctly perform matrix multiplication using CUDA and
print the result.
*/