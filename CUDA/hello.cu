
#include <hip/hip_runtime.h>
#include <cmath>
// #include <iomanip>
#include <cstdio>
#include <iostream>
// #include <ostream>

__global__ void add(int n, float *x, float *y) {
  for (int i = 0; i < n; i++) {
    y[i] += x[i];
  }
}

void fmain(void) {
  // std::cout << "hi CUDA" << std::endl;

  int N = 1 << 20;
  // float *x = new float[N], *y = new float[N];
  float *x, *y;
  hipMallocManaged(&x, N * sizeof(float));
  hipMallocManaged(&y, N * sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f, y[i] = 2.0f;
  }

  // add(N, x, y);
  int second_parameter = 32 * 5;
  printf("%d\n", second_parameter);
  add<<<1, second_parameter>>>(N, x, y);
  hipDeviceSynchronize(); // prbably to avoid race condition

  // Error checking
  float maxError = 0.0f;
  for (int i = 0; i < N; i++) {
    maxError = fmax(fabs(y[i] - 3.0f), maxError);
  }
  // std::cout << std::setprecision(6);
  std::cout << "Max error is : " << maxError << std::endl;

  // delete[] x, delete[] y;
  hipFree(x), hipFree(y);
  //
}

int main(void) {
  int t = 15;
  while (t--) {
    fmain();
    hipDeviceSynchronize();
  }
}